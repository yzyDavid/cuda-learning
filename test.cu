#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <ctime>

#include "hip/hip_runtime.h"

const unsigned int MAX_THREADS_PER_BLOCK = 1024;
const size_t DATA_COUNTS = 50 * 1024 * 1024;

int generated_random_numbers(int *arr, size_t counts)
{
	srand(static_cast<unsigned int>(time(0)));
	for (size_t i = 0; i < counts; i++)
		arr[i] = rand() % 10;
	return 0;
}

void __global__ worker_function(int *data, size_t count, int64_t *result)
{
	int64_t sum = 0;
	for(size_t i = 0; i < count; i++)
		sum += data[i] * data[i];
	*result = sum;
}

void start_work()
{
	auto pData = new int[DATA_COUNTS];
	int *gpuData = nullptr;

	generated_random_numbers(pData, DATA_COUNTS);
	hipMalloc((void **)(&gpuData), sizeof(int) * DATA_COUNTS);
	hipMemcpy(gpuData, pData, sizeof(int) * DATA_COUNTS, hipMemcpyHostToDevice);

	int64_t result;
	int64_t *pgpuResult = nullptr;
	hipMalloc((void **)&pgpuResult, sizeof(int64_t));
	worker_function<<<1, MAX_THREADS_PER_BLOCK, 10240>>>(gpuData, DATA_COUNTS, pgpuResult);
	hipMemcpy(&result, pgpuResult, sizeof(int64_t), hipMemcpyDeviceToHost);
	hipFree(gpuData);
	hipFree(pgpuResult);
	std::cout << "RESULT: " << result << std::endl;

	delete pData;
}

int queryCudaDevicesCount()
{
	int count;

	hipGetDeviceCount(&count);
	if (count == 0)
		return 0;

	return count;
}

int getFirstDeviceProperties()
{
	using namespace std;
	hipDeviceProp_t prop;

	if (hipGetDeviceProperties(&prop, 0) != hipSuccess) return -1;

	printf("\nMajor version of first CUDA device: %d\n", prop.major);
	printf("Minor version of first CUDA device: %d\n", prop.minor);
	printf("Is ECC Enabled: %d\n", prop.ECCEnabled);
	printf("Async Engine Count: %d\n", prop.asyncEngineCount);
	printf("Can map host memory: %d\n", prop.canMapHostMemory);
	printf("clock rate: %d\n", prop.clockRate);
	cout << "Concurrent Kernels: " << prop.concurrentKernels << endl;
	cout << "Global L1 Cache supported: " << prop.globalL1CacheSupported << endl;
	cout << "Max Threads Dimension: " << prop.maxThreadsDim << endl;
	cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << endl;
	cout << "Max Threads per MultiProcessor: " << prop.maxThreadsPerMultiProcessor << endl;
	cout << "MultiProcessorCount: " << prop.multiProcessorCount << endl;
	cout << "Device Name: " << prop.name << endl;
	cout << "Total Global Memory: " << prop.totalGlobalMem << endl;
	cout << endl;
	return 0;
}

int setFirstDevice()
{
	hipSetDevice(0);
	return 0;
}

int main()
{
	std::printf("Numbers of devices support CUDA on this computer: %d\n", queryCudaDevicesCount());
	assert(getFirstDeviceProperties() == 0);
	setFirstDevice();

	start_work();
	return 0;
}

